#include "cuda_rig.h"

#include <algorithm>
#include <cmath>
#include <omp.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

void CudaRig::Init() {
  // Initialize test memory.
  test_init_(mem_);
}

int CudaRig::InitAndCopy(void **dev_mem, void *host_mem, size_t sz) {
  hipError_t status;
  status = hipMalloc(dev_mem, sz);
  checkCudaErrors(status);
  // Copy host memory to the GPU.
  status =
      hipMemcpy(*dev_mem, host_mem, sz, hipMemcpyHostToDevice);
  checkCudaErrors(status);

  return status;
}

void CudaRig::StartCudaTimer(CudaTimer *t) {
  hipError_t status;
  // Create and start timer.
  hipDeviceSynchronize();

  // Allocate CUDA events that we'll use for timing.
  status = hipEventCreate(&(t->start));
  checkCudaErrors(status);
  status = hipEventCreate(&(t->stop));
  checkCudaErrors(status);

  // Record the start event.
  status = hipEventRecord(t->start, NULL);
  checkCudaErrors(status);
}

void CudaRig::StopCudaTimer(CudaTimer *t){
  hipError_t status;
  // Record the stop event.
  status = hipEventRecord(t->stop, NULL);
  checkCudaErrors(status);

  // Wait for the stop event to complete.
  status = hipEventSynchronize(t->stop);
  checkCudaErrors(status);
}